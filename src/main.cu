#include <iostream>
#include <cstdlib>
#include <math.h>
#include <random>
#include <chrono>

#include </usr/local/cuda/include/hip/hip_runtime.h>
#include </usr/local/cuda/include/hiprand.h>
#include </usr/local/cuda/include/hiprand/hiprand_kernel.h>

#include "../include/errorHandling.h"
#include "../include/ptrKernel.h"

// Function that computes the minimum value of two input
#define imin(a,b) (a<b?a:b)

int main(){

    // Number of particles
    size_t nParticles{10};

    // Size of the Box
    size_t sizeBox{50};

    // Particles radius
    double radio{1.5};

    // Pointers to allocate coordinates of particles on host
    double *xCoords{nullptr}, *yCoords{nullptr}, *angleCoords{nullptr};

    // Pointer to allocate the distances between particles
    double *particleDistance{nullptr};

    // Allocate memory on the CPU
    xCoords     = new double[nParticles];
    yCoords     = new double[nParticles];
    angleCoords = new double[nParticles];

    particleDistance    = new double[nParticles * nParticles];


    int initConditions = confNonOverlapingInitialConditions(nParticles, sizeBox, radio, xCoords, yCoords, angleCoords);

    if(initConditions == 1){
        // Clear memory if we can´t arrange the particles
        delete[] xCoords;       xCoords = nullptr;
        delete[] yCoords;       yCoords = nullptr;
        delete[] angleCoords;   angleCoords = nullptr;

        delete[] particleDistance; particleDistance = nullptr;

        return 1;
    }
    else{
        std::cout << "The initial condition was setted :D" << std::endl;
    }

    // display positions and angles
    std::cout << "X POSITIONS" << std::endl;
    displayPtr(nParticles, xCoords);

    std::cout << "Y POSITIONS" << std::endl;
    displayPtr(nParticles, yCoords);

    std::cout << "ANGLES" << std::endl;
    displayPtr(nParticles, angleCoords);



    delete[] xCoords;       xCoords = nullptr;
    delete[] yCoords;       yCoords = nullptr;
    delete[] angleCoords;   angleCoords = nullptr;

    delete[] particleDistance; particleDistance = nullptr;
    return 0;
}