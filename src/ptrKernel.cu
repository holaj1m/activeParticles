#include "hip/hip_runtime.h"
#include <math.h>
#include <random>
#include <chrono>
#include <iostream>

#include "../include/ptrKernel.h"

int confNonOverlapingInitialConditions(size_t nParticles, size_t sizeBox, double radio, double *ptrX, double *ptrY, double *ptrAngle){
    // Take current time as seed
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);

    // Length of the box 
    double boxLength{sizeBox*0.5};

    // Overlaping distance
    double overlapingDistance{4*radio*radio};

    // Uniform distribution between 0 and 2PI
    std::uniform_real_distribution<double> angleDistribution(0.0, 2*M_PI);
    // Uniform distribution between -sizeBox and sizeBox
    std::uniform_real_distribution<double> positionDistribution(-boxLength, boxLength);

    // Number of max attemps to set the initial condition
    size_t tolerance{2 * nParticles};

    // Proposed positons and distance
    double proposedX{}, proposedY{}, distance{};

    for(size_t particle{}; particle < nParticles; particle++){
        // Variable to count the attempts placing a particle
        size_t attempt{0};

        // Variable to manage errors
        bool placed{false};

        while(attempt < tolerance){

            // Assign positions for a particle
            proposedX = positionDistribution(generator);
            proposedY = positionDistribution(generator);

            // Assuming that particles does not overlap each other
            bool overlap{false};

            // Compute the distance with particles already assigned
            for(size_t previousParticles{}; previousParticles < particle; previousParticles++){
                distance = (pow((proposedX - ptrX[previousParticles]),2) + pow((proposedY - ptrY[previousParticles]),2));

                // If particles are overlaping change the value of overlap and break the loop
                if(distance < overlapingDistance){
                    overlap = true;
                    break;
                }
            }

            if(!overlap){
                ptrX[particle]      = proposedX;
                ptrY[particle]      = proposedY;
                ptrAngle[particle]  = angleDistribution(generator);

                placed = true;

                break;
            }
            else{
                // Increment attempt
                attempt++;
            }
        }
        
        // Return 1 if any particle could not be placed
        if (!placed) {
            std::cerr << "We coudn't place the particle " << particle << " after " << tolerance << " tries.\n";
            return 1; 
        }

    }

    return 0;
}


// Display pointer
void displayPtr(size_t ptrSize, double *ptr){
    std::cout << "[" << ptr[0] << ", ";
    for(size_t i{1}; i < ptrSize - 1; i++){
        std::cout << ptr[i] << ", ";
    }
    std::cout << ptr[ptrSize-1] << "]" << std::endl;
}

